#include "hip/hip_runtime.h"
#include "common.h"
#include "reference.h"

const int testM = 5120;
const int testN = 4096;
const int testK = 2048;

static constexpr int cluster_M = 2;
static constexpr int cluster_N = 1;
static constexpr int wg_number = 3;

static constexpr int blockM = 128;
static constexpr int blockN = 128;
static constexpr int blockK = 64;
static constexpr int stages = 7;

namespace utils {

using tmaDescriptor = CUtensorMap;

template <class T>
inline CUtensorMapDataType to_CUtensorMapDataType() {
    if constexpr (std::is_same<T, int8_t>::value) {
        return CU_TENSOR_MAP_DATA_TYPE_UINT8;
    } else if constexpr (std::is_same<T, uint8_t>::value) {
        return CU_TENSOR_MAP_DATA_TYPE_UINT8;
    } else if constexpr (std::is_same<T, uint16_t>::value) {
        return CU_TENSOR_MAP_DATA_TYPE_UINT16;
    } else if constexpr (std::is_same<T, uint32_t>::value) {
        return CU_TENSOR_MAP_DATA_TYPE_UINT32;
    } else if constexpr (std::is_same<T, uint64_t>::value) {
        return CU_TENSOR_MAP_DATA_TYPE_UINT64;
    } else if constexpr (std::is_same<T, int32_t>::value) {
        return CU_TENSOR_MAP_DATA_TYPE_INT32;
    } else if constexpr (std::is_same<T, int64_t>::value) {
        return CU_TENSOR_MAP_DATA_TYPE_INT64;
    } else if constexpr (std::is_same<T, half_t>::value) {
        return CU_TENSOR_MAP_DATA_TYPE_FLOAT16;
    } else if constexpr (std::is_same<T, float>::value) {
        return CU_TENSOR_MAP_DATA_TYPE_FLOAT32;
    } else if constexpr (std::is_same<T, double>::value) {
        return CU_TENSOR_MAP_DATA_TYPE_FLOAT64;
    } else if constexpr (std::is_same<T, bfloat16>::value) {
        return CU_TENSOR_MAP_DATA_TYPE_BFLOAT16;
    } else if constexpr (std::is_same<T, tfloat32_t>::value) {
        return CU_TENSOR_MAP_DATA_TYPE_TFLOAT32;
    } else {
        static_assert(sizeof(T) < 0, "Unknown TMA Format!");
    }
}

enum class SmemSwizzleBits : uint8_t {
    DISABLE = 0,
    B32 = 1,
    B64 = 2,
    B128 = 3,
};

template <int B, int M, int S>
HOST_DEVICE constexpr SmemSwizzleBits get_tma_swizzle_bits(Swizzle<B, M, S>) {
    if constexpr (M==4) {
        switch (B) {
            default:
            static assert(0 <=B && B <=3, "expected B = 0,1,2 or 3 when M==4. Unsupported layout swizzle.");
            case 3:
                return SmemSwizzleBits::B128;
            case 2:
                return SmemSwizzleBits::B64;
            case 1:
                return SmemSwizzleBits::B32;
            case 0:
                return SmemSwizzleBits::DISABLE;


        }

    }
} else {
    static_assert(M < 0, "unsupported layout swizzle.");
}
}


inline CUtensorMapSwizzle to_CUtensorMapSwizzle(SmemSwizzleBits const& t) {
  switch (t) {
    default:
      assert(false && "Unknown SmemSwizzleBits!");
    case SmemSwizzleBits::DISABLE:
      return CU_TENSOR_MAP_SWIZZLE_NONE;
    case SmemSwizzleBits::B32:
      return CU_TENSOR_MAP_SWIZZLE_32B;
    case SmemSwizzleBits::B64:
      return CU_TENSOR_MAP_SWIZZLE_64B;
    case SmemSwizzleBits::B128:
      return CU_TENSOR_MAP_SWIZZLE_128B;
  }
}


}// minor dims move faster than major
template <int BlockMajorSize, int BlockMinorSize, int B, int M, int S>
TmaDescriptor make_tma_copy_desc(DType* gmem_ptr, int shape_major,
                                int shape_minor,
                                Swizzle<B,M,S> const& swizzle,
                                uint32_t num_multicast) {
void* gmem_address = (void*)gmem_ptr;
uint64_t gmem_prob_shape[5] = {(uint64_t) shape_minor, (uint64_t)shape_major, 1,1,1};
uint64)t gmem_prob_stride[5] = {sizeof(DType), sizeof(DType) * shape_minor, 0, 0,0};


  assert((reinterpret_cast<uint64_t>(gmem_address) & 0b1111) == 0);
  assert(gmem_prob_shape[0] >= (uint64_t(1)));
  assert(gmem_prob_shape[0] <= (uint64_t(1) << 32));
  assert(gmem_prob_shape[1] >= (uint64_t(1)));
  assert(gmem_prob_shape[1] <= (uint64_t(1) << 32));
  assert(gmem_prob_shape[2] >= (uint64_t(1)));
  assert(gmem_prob_shape[2] <= (uint64_t(1) << 32));
  assert(gmem_prob_shape[3] >= (uint64_t(1)));
  assert(gmem_prob_shape[3] <= (uint64_t(1) << 32));
  assert(gmem_prob_shape[4] >= (uint64_t(1)));
  assert(gmem_prob_shape[4] <= (uint64_t(1) << 32));

  assert(gmem_prob_stride[0] == sizeof(DType));
  assert(gmem_prob_stride[1] < (uint64_t(1) << 40));
  assert((gmem_prob_stride[1] & 0b1111) == 0);
  assert(gmem_prob_stride[2] < (uint64_t(1) << 40));
  assert((gmem_prob_stride[2] & 0b1111) == 0);
  assert(gmem_prob_stride[3] < (uint64_t(1) << 40));
  assert((gmem_prob_stride[3] & 0b1111) == 0);
  assert(gmem_prob_stride[4] < (uint64_t(1) << 40));
  assert((gmem_prob_stride[4] & 0b1111) == 0);

  assert(BlockMajorSize % num_multicast == 0);

  uint32_t smem_box_shape[5] = {uint32_t(BlockMinorSize),
                                uint32_t(BlockMajorSize / num_multicast), 1,1,1};
    uint32_t smem_box_stride[5] = {1,1,1,1,1};


  assert(smem_box_shape[0] >= (uint32_t(1)));  // Size must be min 1
  assert(smem_box_shape[0] <=
         (uint32_t(1) << 8));                  // Size must be max 2^8 = 256
  assert(smem_box_shape[1] >= (uint32_t(1)));  // Size must be min 1
  assert(smem_box_shape[1] <=
         (uint32_t(1) << 8));                  // Size must be max 2^8 = 256
  assert(smem_box_shape[2] >= (uint32_t(1)));  // Size must be min 1
  assert(smem_box_shape[2] <=
         (uint32_t(1) << 8));                  // Size must be max 2^8 = 256
  assert(smem_box_shape[3] >= (uint32_t(1)));  // Size must be min 1
  assert(smem_box_shape[3] <=
         (uint32_t(1) << 8));                  // Size must be max 2^8 = 256
  assert(smem_box_shape[4] >= (uint32_t(1)));  // Size must be min 1
  assert(smem_box_shape[4] <=
         (uint32_t(1) << 8));  // Size must be max 2^8 = 256

  assert(smem_box_stride[0] >= (uint32_t(1)));  // Stride must be min 1
  assert(smem_box_stride[0] <= (uint32_t(8)));  // Stride must be max 2^3 = 8
  assert(smem_box_stride[1] >= (uint32_t(1)));  // Stride must be min 1
  assert(smem_box_stride[1] <= (uint32_t(8)));  // Stride must be max 2^3 = 8
  assert(smem_box_stride[2] >= (uint32_t(1)));  // Stride must be min 1
  assert(smem_box_stride[2] <= (uint32_t(8)));  // Stride must be max 2^3 = 8
  assert(smem_box_stride[3] >= (uint32_t(1)));  // Stride must be min 1
  assert(smem_box_stride[3] <= (uint32_t(8)));  // Stride must be max 2^3 = 8
  assert(smem_box_stride[4] >= (uint32_t(1)));  // Stride must be min 1
  assert(smem_box_stride[4] <= (uint32_t(8)));  // Stride must be max 2^3 = 8

TmaDescriptor tma_desc = {0};
CUtensorMapDataType tma_format = to_CUtensorMapDataType<typename std::remove_cv<DType>::type>();
CUtensorMapInterleave tma_interleave = CU_TENSOR_MAP_INTERLEAVE_NONE;
CUtensorMapL2promotion tma_l2Promotion = CU_TENSOR_MAP_L2_PROMOTION_L2_128B;
CUtensorMapFloatOOBfill tma_oobFill = CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE;

CUtensorMapSwizzle smem_swizzle = to_CUtensorMapSwizzle(get_tma_swizzle_bits(swizzle));
hipError_t result = cuTensorMapEncodeTiled(&tma_desc, tma_format, TmaDim, gmem_address, gmem_prob_shape,
gmem_prob_stride+1, smem_box_shape, smem_box_stride, tma_interleav,
smem_swizzle, tma_l2Promotion, tma_oobFill);

if (result != hipSuccess) {
    std::cerr << "TMA Desc Addr:   " << &tma_desc << "\nformat         "
              << tma_format << "\ndim            " << TmaDim
              << "\ngmem_address   " << gmem_address << "\nglobalDim      "
              << gmem_prob_shape << "\nglobalStrides  " << gmem_prob_stride
              << "\nboxDim         " << smem_box_shape << "\nelementStrides "
              << smem_box_stride << "\ninterleave     " << tma_interleave
              << "\nswizzle        " << smem_swizzle << "\nl2Promotion    "
              << tma_l2Promotion << "\noobFill        " << tma_oobFill
              << std::endl;
    std::cerr << "Error: Failed to initialize the TMA descriptor " << result
              << std::endl;
    assert(false);
  }
    return tma_desc;


                                }


}
